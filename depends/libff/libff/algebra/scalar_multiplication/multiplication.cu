#include "hip/hip_runtime.h"
#define GRID_SIZE 32
#define BLOCK_SIZE 16 

template<typename FieldT> 
__device__ __constant__ FieldT zero;

template <typename T, unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template<typename T>
__device__ T out;

template <typename T, typename FieldT, unsigned int blockSize>
__global__ void cuda_multi_exp_inner(T *vec, FieldT *scalar, unsigned int field_size) {
    extern __shared__ T sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockSize*2) + tid;
    unsigned int gridSize = blockSize*2*gridDim.x;

    sdata[tid] = zero;
    while (i < field_size) { 
        sdata[tid] += scalar[i] * vec[i]; i += gridSize; }
    __syncthreads();
    
    if (blockSize >= 2048) { if (tid < 1024) { sdata[tid] += sdata[tid + 1024]; } __syncthreads(); }
    if (blockSize >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); }
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

    if (tid < 32) warpReduce(sdata, tid);
    if (tid == 0) out = sdata[0];
}

template<typename T, typename FieldT, multi_exp_method Method,
    typename std::enable_if<(Method == multi_exp_method_cuda), int>::type = 0>
T multi_exp_inner(
    typename std::vector<T>::const_iterator vec_start,
    typename std::vector<T>::const_iterator vec_end,
    typename std::vector<FieldT>::const_iterator scalar_start,
    typename std::vector<FieldT>::const_iterator scalar_end)
{
    T *d_vec;
    FieldT *d_scalar
    size_t vec_size = (vec_end - vec_start) * sizeof(T);
    size_t scalar_size = (scalar_end - scalar_start) * sizeof(FieldT);

    hipMalloc((void **)&d_vec, vec_size);
    hipMalloc((void **)&d_scalar, scalar_size;
	
    hipMemcpy(d_vec, &vec_start, vec_size, hipMemcpyHostToDevice);
    hipMemcpy(d_scalar, &scalar_start, scalar_size, hipMemcpyHostToDevice);
    smemSize = vec_size / 2;	

    dim3 dimGrid (GRID_SIZE, GRID_SIZE);
    dim3 dimBlock (BLOCK_SIZE, BLOCK_SIZE);
    threads = dimGrid.x * dimGrid.y;
    switch (threads)
    {
        case 2048:
        cuda_multi_exp_inner<2048><<< dimGrid, dimBlock, smemSize >>>(d_vec, d_scalar); break;
        case 1024:
        cuda_multi_exp_inner<1024><<< dimGrid, dimBlock, smemSize >>>(d_vec, d_scalar); break;
        case 512:
        cuda_multi_exp_inner<512><<< dimGrid, dimBlock, smemSize >>>(d_vec, d_scalar); break;
        case 256:
        cuda_multi_exp_inner<256><<< dimGrid, dimBlock, smemSize >>>(d_vec, d_scalar); break;
    }

    T* res; T result;
    CUDA_CALL (hipGetSymbolAddress((void**) &res, out<T>));
    hipMemcpy(result, res, sizeof(T), hipMemcpyDeviceToHost);

    hipFree(d_vec);
    hipFree(d_scalar);
    return result;
}