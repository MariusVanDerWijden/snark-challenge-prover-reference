#include "hip/hip_runtime.h"
/*****************************************************************************
 Implementation of Fast Fourier Transformation on Finite Elements
 *****************************************************************************
 * @author     Marius van der Wijden
 * Copyright [2019] [Marius van der Wijden]
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <vector>
#include <iostream>
#include "fft_kernel.h"
#include <libff/algebra/fields/field_utils.hpp>
#include <libff/common/utils.hpp>
#include <libff/algebra/curves/mnt753/mnt4753/mnt4753_pp.hpp>
#include <libff/algebra/curves/mnt753/mnt6753/mnt6753_pp.hpp>
#include <libff/algebra/scalar_multiplication/multiexp.hpp>
#include <libff/common/profiling.hpp>
#include <libff/common/rng.hpp>
#include <libff/common/utils.hpp>
#include <libff>
#include <libff/algebra/fields>

#define LOG_NUM_THREADS 16
#define NUM_THREADS (1 << LOG_NUM_THREADS)
#define LOG_CONSTRAINTS 18
#define CONSTRAINTS (1 << LOG_CONSTRAINTS)

#define CUDA_CALL( call )               \
{                                       \
hipError_t result = call;              \
if ( hipSuccess != result )            \
    std::cerr << "CUDA error " << result << " in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString( result ) << " (" << #call << ")" << std::endl;  \
}

__device__ __forceinline__
size_t bitreverse(size_t n, const size_t l)
{
    size_t r = 0;
    for (size_t k = 0; k < l; ++k)
    {
        r = (r << 1) | (n & 1);
        n >>= 1;
    }
    return r;
}

template<typename FieldT> 
__device__ __constant__ FieldT omega;
template<typename FieldT> 
__device__ __constant__ FieldT one;
template<typename FieldT> 
__device__ __constant__ FieldT zero;
template<typename FieldT>
__device__ FieldT field[CONSTRAINTS];
template<typename FieldT>
__device__ FieldT out[CONSTRAINTS];

template<typename FieldT>  __global__ void cuda_fft()
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t log_m = LOG_CONSTRAINTS;
    const size_t length = CONSTRAINTS;
    const size_t block_length = 1ul << (log_m - LOG_NUM_THREADS) ;
    const size_t startidx = idx * block_length;
    assert (CONSTRAINTS == 1ul<<log_m);
    if(startidx > length)
        return;
    FieldT a [block_length];
    //zero()
    memset(a, block_length,  0); 

    //TODO algorithm is non-deterministic because of padding
    set_mod(omega<FieldT>);
    FieldT omega_j = omega<FieldT>;
    pow(omega_j, idx);
    FieldT omega_step = omega<FieldT>;
    pow(omega_step, idx << (log_m - LOG_NUM_THREADS));
    
    FieldT elt = one<FieldT>;
    for (size_t i = 0; i < 1ul<<(log_m - LOG_NUM_THREADS); ++i)
    {
        for (size_t s = 0; s < NUM_THREADS; ++s)
        {
            // invariant: elt is omega^(j*idx)
            size_t id = (i + (s<<(log_m - LOG_NUM_THREADS))) % (1u << log_m);
            FieldT tmp = field<FieldT>[id];
            mul(tmp, elt);
            add(a[i], tmp);
            //a[i] += field<FieldT>[id] * elt;
            mul(elt, omega_step);
            //elt *= omega_step;
        }
        mul(elt, omega_j);
        //elt *= omega_j;
    }

    FieldT omega_num_cpus = omega<FieldT>;
    pow(omega_num_cpus, NUM_THREADS);
    //const FieldT omega_num_cpus = omega<FieldT> ^ NUM_THREADS;
    
    //Do not remove log2f(n), otherwise register overflow
    size_t n = block_length, logn = log2f(n);
    assert (n == (1u << logn));

    /* swapping in place (from Storer's book) */
    for (size_t k = 0; k < n; ++k)
    {
        const size_t rk = bitreverse(k, logn);
        if (k < rk)
        {
            FieldT tmp = a[k];
            a[k] = a[rk];
            a[rk] = tmp;
        }
    }

    size_t m = 1; // invariant: m = 2^{s-1}
    for (size_t s = 1; s <= logn; ++s)
    {
        // w_m is 2^s-th root of unity now
        FieldT w_m = omega_num_cpus;
        pow(w_m, n/2*m);
        //const FieldT w_m = omega_num_cpus^(n/(2*m));

        for (size_t k = 0; k < n; k += 2*m)
        {
            FieldT w = one<FieldT>;
            for (size_t j = 0; j < m; ++j)
            {
                FieldT t = w;
                mul(t, a[k+j+m]);
                //const FieldT t = w * a[k+j+m];
                FieldT tmp = a[k+j];
                subtract(tmp, t);
                a[k+j+m] = tmp;
                //a[k+j+m] = a[k+j] - t;
                add(a[k+j], t);
                //a[k+j] += t;
                mul(w, w_m);
                //w *= w_m;
            }
        }
        m = m << 1;
    }
    for (size_t j = 0; j < 1ul<<(log_m - LOG_NUM_THREADS); ++j)
    {
        if(((j << LOG_NUM_THREADS) + idx) < length)
            out<FieldT>[(j<<LOG_NUM_THREADS) + idx] = a[j];
    }
}

template<typename n, typename mod>
void best_fft(std::vector<libff::Fp_model<n, mod>> &v, const libff::Fp_model<n, mod> &omg);
//void best_fft (FieldT *a, size_t _size, const FieldT &omg)
{
	int cnt;
    hipGetDeviceCount(&cnt);
    printf("CUDA Devices: %d, Field size: %d, Field count: %d\n", cnt, sizeof(FieldT), a.size());
    assert(a.size() == CONSTRAINTS);

    CUDA_CALL( hipMemcpyToSymbol(HIP_SYMBOL(field<FieldT>), &a[0], CONSTRAINTS, hipMemcpyHostToDevice) );
    
    const FieldT oneElem = FieldT::one();
    const FieldT zeroElem = FieldT::zero();
    CUDA_CALL( hipMemcpyToSymbol(HIP_SYMBOL(omega<FieldT>), &omg, sizeof(FieldT), 0, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpyToSymbol(HIP_SYMBOL(one<FieldT>), &oneElem, sizeof(FieldT), 0, hipMemcpyHostToDevice) );
    CUDA_CALL( hipMemcpyToSymbol(HIP_SYMBOL(zero<FieldT>), &zeroElem, sizeof(FieldT), 0, hipMemcpyHostToDevice) );

    size_t blocks = NUM_THREADS / 1024 + 1;
    size_t threads = NUM_THREADS > 1024 ? 1024 : NUM_THREADS;
    printf("threads %d, blocks %d, threads %d \n",NUM_THREADS, blocks, threads);
    cuda_fft<FieldT> <<<blocks,threads>>>();
        
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        // print the CUDA error message and exit
        printf("CUDA error: %s\n", hipGetErrorString(error));
        exit(-1);
    }
    CUDA_CALL( hipDeviceSynchronize();)
    
    FieldT* res;
    CUDA_CALL (hipGetSymbolAddress((void**) &res, out<FieldT>));

    FieldT * result = (FieldT*) malloc (sizeof(FieldT) * a.size());
    hipMemcpy(result, res, sizeof(FieldT) * a.size(), hipMemcpyDeviceToHost);

    //std::copy(result, result + _size, a);
    CUDA_CALL( hipDeviceSynchronize();)
}

//List with all templates that should be generated
//template void best_fft(std::vector<int> &a, const int &omg);

